
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <chrono>
#include <fstream>
#include <iostream>

using namespace std;

#define kmean 100
#define H 1000
#define W 1000
#define iterations 200

__device__ float sqDistance(float *p1, float *p2) {
  return (p1[0] - p2[0]) * (p1[0] - p2[0]) + (p1[1] - p2[1]) * (p1[1] - p2[1]) +
         (p1[2] - p2[2]) * (p1[2] - p2[2]);
}

__global__ void kmeanKernel(float *pixels, float *cluster_pos,
                            float *sorteddata_pixels, float *cluster_meanpoint,
                            int *cluster_hitcount, int repetition) {

  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if (i < H * W * 3) {
    if (i % 3 == 0) {

      float distance;
      float bestdistance;
      int bestcluster = 0;

      distance = INFINITY;
      bestdistance = distance;

      for (int m = 0; m < 3 * kmean; m = m + 3) {
        distance = sqDistance(cluster_pos + m, pixels + i);
        if (distance < bestdistance) {
          bestcluster = m / 3;
          bestdistance = distance;
        }
      }

      if (repetition == (iterations - 1)) {
        sorteddata_pixels[i] = cluster_pos[bestcluster * 3];
        sorteddata_pixels[i + 1] = cluster_pos[bestcluster * 3 + 1];
        sorteddata_pixels[i + 2] = cluster_pos[bestcluster * 3 + 2];
      }

      // __syncthreads();
      atomicAdd(&cluster_meanpoint[bestcluster * 3], pixels[i]);
      atomicAdd(&cluster_meanpoint[bestcluster * 3 + 1], pixels[i + 1]);
      atomicAdd(&cluster_meanpoint[bestcluster * 3 + 2], pixels[i + 2]);
      atomicAdd(&cluster_hitcount[bestcluster], 1);
    }
  }
}

__global__ void updateMeans(float *cluster_pos, float *pixels,
                            int *cluster_hitcount, float *meanpoint) {
  // The position is now the mean and the other parameters are restored
  int m = threadIdx.x + blockDim.x * blockIdx.x;

  if (m < 3 * kmean) {
    if (m % 3 == 0) {
      if (cluster_hitcount[m / 3] == 0) {
        // If no hit, then assign random value again
        cluster_pos[m] = pixels[m];
        cluster_pos[m + 1] = pixels[m + 1];
        cluster_pos[m + 2] = pixels[m + 2];
      }
      // Doing the mean of the meanpoint which contains the sum of all the hit
      // coordinates
      cluster_pos[m] = meanpoint[m] / (cluster_hitcount[m / 3]);
      cluster_pos[m + 1] = meanpoint[m + 1] / (cluster_hitcount[m / 3]);
      cluster_pos[m + 2] = meanpoint[m + 2] / (cluster_hitcount[m / 3]);

      // Restoring hit and meanpoint
      meanpoint[m] = 0;
      meanpoint[m + 1] = 0;
      meanpoint[m + 2] = 0;
      cluster_hitcount[m / 3] = 0;
    }
  }
}

int main() {
  ifstream inputValues("image.txt");

  float *host_pixels = 0;
  float *host_cluster_pos = 0;
  float *host_sorteddata_pixels = 0;
  float *host_meanpoint = 0;
  int *host_cluster_hitcount = 0;

  float *device_pixels = 0;
  float *device_cluster_pos = 0;
  float *device_sorteddata_pixels = 0;
  float *device_meanpoint = 0;
  int *device_cluster_hitcount = 0;

  host_pixels = (float *)malloc(3 * H * W * sizeof(float));
  host_cluster_pos = (float *)malloc(3 * kmean * sizeof(float));
  host_sorteddata_pixels = (float *)malloc(3 * H * W * sizeof(float));
  host_meanpoint = (float *)malloc(3 * kmean * sizeof(float));
  host_cluster_hitcount = (int *)malloc(kmean * sizeof(int));

  hipMalloc((void **)&device_pixels, 3 * H * W * sizeof(float));
  hipMalloc((void **)&device_cluster_pos, 3 * kmean * sizeof(float));
  hipMalloc((void **)&device_sorteddata_pixels, 3 * H * W * sizeof(float));
  hipMalloc((void **)&device_meanpoint, 3 * kmean * sizeof(float));
  hipMalloc((void **)&device_cluster_hitcount, kmean * sizeof(int));

  for (int i = 0; i < 3 * H * W; ++i) {
    inputValues >> host_pixels[i];
  }

  inputValues.close();

  auto start = chrono::high_resolution_clock::now();

  // Asign random values to start the iteration
  for (int i = 0; i < 3 * kmean; i = i + 3) {
    host_cluster_pos[i] = host_pixels[i];
    host_cluster_pos[i + 1] = host_pixels[i + 1];
    host_cluster_pos[i + 2] = host_pixels[i + 2];
  }

  int block_size = 1024;
  int blocks_no = 3 * H * W / block_size;

  if (3 * H * W % block_size)
    ++blocks_no;

  int blocks_no2 = 3 * kmean / block_size;

  if (3 * kmean % block_size)
    ++blocks_no2;

  int repetition = 0;

  // printf("Blocks no, Blocks size: %d, %d\n", blocks_no, block_size);
  // printf("Blocks no, Blocks size: %d, %d\n", blocks_no2, block_size);

  hipMemcpy(device_pixels, host_pixels, 3 * H * W * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(device_cluster_pos, host_cluster_pos, 3 * kmean * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(device_meanpoint, host_meanpoint, 3 * kmean * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(device_cluster_hitcount, host_cluster_hitcount,
             kmean * sizeof(int), hipMemcpyHostToDevice);

  while (repetition < iterations) {
    repetition++;

    kmeanKernel<<<blocks_no, block_size>>>(
        device_pixels, device_cluster_pos, device_sorteddata_pixels,
        device_meanpoint, device_cluster_hitcount, repetition);

    hipDeviceSynchronize();
    // printf("\n Error msg: %s \n", cudaGetErrorString(cudaGetLastError()));

    updateMeans<<<blocks_no2, block_size>>>(device_cluster_pos, device_pixels,
                                            device_cluster_hitcount,
                                            device_meanpoint);

    hipDeviceSynchronize();
    // printf("\n Error msg: %s \n", cudaGetErrorString(cudaGetLastError()));
  }

  hipMemcpy(host_sorteddata_pixels, device_sorteddata_pixels,
             3 * H * W * sizeof(float), hipMemcpyDeviceToHost);

  // for (int i = 0; i < 3 * H * W; i = i + 3) {
  //   printf("%1.6f\t%1.6f\t%1.6f\n", host_sorteddata_pixels[i],
  //          host_sorteddata_pixels[i + 1], host_sorteddata_pixels[i + 2]);
  // }

  free(host_pixels);
  free(host_cluster_pos);
  free(host_sorteddata_pixels);
  free(host_meanpoint);
  free(host_cluster_hitcount);

  hipFree(device_pixels);
  hipFree(device_cluster_pos);
  hipFree(device_sorteddata_pixels);
  hipFree(device_meanpoint);
  hipFree(device_cluster_hitcount);

  auto stop = chrono::high_resolution_clock::now();

  cout << "k means done in "
       << float(chrono::duration_cast<chrono::milliseconds>(stop - start)
                    .count() /
                1000.0)
       << " s" << endl;
}
